
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>

#define THREADS 512
#ifdef __cplusplus
extern "C"
{
#endif

__global__ void bitonicSort(float *d_inputArray, int blockSize, int strideLength, int number_of_elements)
{
  int index = blockIdx.x*blockDim.x + threadIdx.x;

  if(index >= number_of_elements)
    return;
  
  int blockNumber = index/blockSize;
  int blockStartAddress = blockNumber*blockSize;

  //computation is performed on only selected indices
  //as according to bitonic sort algorithm
  if( ((index-blockStartAddress)/strideLength) % 2 != 0 )
    return;
  
  //even numbered block will be sorted in increasing order
  //while odd numbered block is sorted in decreasing order
  if(blockNumber % 2 == 0)
  {
    if(d_inputArray[index] > d_inputArray[index + strideLength])
    {
      float temp = d_inputArray[index];
      d_inputArray[index] = d_inputArray[index + strideLength];
      d_inputArray[index + strideLength] = temp;
    }
  }
  else
  {
    if(d_inputArray[index] < d_inputArray[index + strideLength])
    {
      float temp = d_inputArray[index];
      d_inputArray[index] = d_inputArray[index + strideLength];
      d_inputArray[index + strideLength] = temp;
    }
  }
}

int cuda_sort(int number_of_elements, float *a)
{
  //allocating memory on GPU device and copying data from host to GPU device 
 	float *d_inputArray;
	if(!hipMalloc(&d_inputArray, sizeof(float) * number_of_elements) == hipSuccess)
		printf("error in allocating d_inputArray\n");
  
  if(!hipMemcpy(d_inputArray, a, sizeof(float) * number_of_elements, hipMemcpyHostToDevice) == hipSuccess)
    printf("error in copying d_inputArray\n");

	//iterating through input array block by block
	for (int blockSize = 2; blockSize <= number_of_elements; blockSize = blockSize*2)
	{
    //iterating through each block with differnt strideLength
    for(int strideLength = blockSize/2; strideLength >= 1; strideLength = strideLength/2)
    {
      bitonicSort<<<(number_of_elements/1024 + 1),1024>>>(d_inputArray, blockSize, strideLength, number_of_elements);
    }
  }

  //copying data back from GPU device to host memory
  if(!hipMemcpy(a, d_inputArray, sizeof(float) * number_of_elements, hipMemcpyDeviceToHost) == hipSuccess)
    printf("error in copying d_inputArray from device to host\n");
  
  return 0;
}

#ifdef __cplusplus
}
#endif
